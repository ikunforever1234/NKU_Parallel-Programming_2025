#include "hip/hip_runtime.h"
#include "PCFG.h"
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include <vector>
#include <string>
#include <cstring>
using namespace std;

void PriorityQueue::CalProb(PT &pt)
{
    // 计算PriorityQueue里面一个PT的流程如下：
    // 1. 首先需要计算一个PT本身的概率。例如，L6S1的概率为0.15
    // 2. 需要注意的是，Queue里面的PT不是“纯粹的”PT，而是除了最后一个segment以外，全部被value实例化的PT
    // 3. 所以，对于L6S1而言，其在Queue里面的实际PT可能是123456S1，其中“123456”为L6的一个具体value。
    // 4. 这个时候就需要计算123456在L6中出现的概率了。假设123456在所有L6 segment中的概率为0.1，那么123456S1的概率就是0.1*0.15

    // 计算一个PT本身的概率。后续所有具体segment value的概率，直接累乘在这个初始概率值上
    pt.prob = pt.preterm_prob;

    // index: 标注当前segment在PT中的位置
    int index = 0;


    for (int idx : pt.curr_indices)
    {
        // pt.content[index].PrintSeg();
        if (pt.content[index].type == 1)
        {
            // 下面这行代码的意义：
            // pt.content[index]：目前需要计算概率的segment
            // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
            // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
            // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
            // cout << m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.letters[m.FindLetter(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
            // cout << m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.digits[m.FindDigit(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].total_freq << endl;
        }
        index += 1;
    }
    // cout << pt.prob << endl;
}

void PriorityQueue::init()
{
    // cout << m.ordered_pts.size() << endl;
    // 用所有可能的PT，按概率降序填满整个优先队列
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                // 下面这行代码的意义：
                // max_indices用来表示PT中各个segment的可能数目。例如，L6S1中，假设模型统计到了100个L6，那么L6对应的最大下标就是99
                // （但由于后面采用了"<"的比较关系，所以其实max_indices[0]=100）
                // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
                // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
                // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;
        // pt.PrintPT();
        // cout << " " << m.preterm_freq[m.FindPT(pt)] << " " << m.total_preterm << " " << pt.preterm_prob << endl;

        // 计算当前pt的概率
        CalProb(pt);
        // 将PT放入优先队列
        priority.emplace_back(pt);
    }
    // cout << "priority size:" << priority.size() << endl;
}


// 这个函数你就算看不懂，对并行算法的实现影响也不大
// 当然如果你想做一个基于多优先队列的并行算法，可能得稍微看一看了
vector<PT> PT::NewPTs()
{
    // 存储生成的新PT
    vector<PT> res;

    // 假如这个PT只有一个segment
    // 那么这个segment的所有value在出队前就已经被遍历完毕，并作为猜测输出
    // 因此，所有这个PT可能对应的口令猜测已经遍历完成，无需生成新的PT
    if (content.size() == 1)
    {
        return res;
    }
    else
    {
        // 最初的pivot值。我们将更改位置下标大于等于这个pivot值的segment的值（最后一个segment除外），并且一次只更改一个segment
        // 上面这句话里是不是有没看懂的地方？接着往下看你应该会更明白
        int init_pivot = pivot;

        // 开始遍历所有位置值大于等于init_pivot值的segment
        // 注意i < curr_indices.size() - 1，也就是除去了最后一个segment（这个segment的赋值预留给并行环节）
        for (int i = pivot; i < curr_indices.size() - 1; i += 1)
        {
            // curr_indices: 标记各segment目前的value在模型里对应的下标
            curr_indices[i] += 1;

            // max_indices：标记各segment在模型中一共有多少个value
            if (curr_indices[i] < max_indices[i])
            {
                // 更新pivot值
                pivot = i;
                res.emplace_back(*this);
            }

            // 这个步骤对于你理解pivot的作用、新PT生成的过程而言，至关重要
            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }

    return res;
}


#define MAX_VALUE_LEN 32
#define MAX_PTS_BATCH 64
void PriorityQueue::PopNext() 
{
    int B = min((int)priority.size(), MAX_PTS_BATCH);
    vector<PT> batch(priority.begin(), priority.begin() + B);
    priority.erase(priority.begin(), priority.begin() + B);
    
    for (auto &pt : batch) 
    {
        Generate(pt);
    }
    
    for (auto &pt : batch) 
    {
        auto newpts = pt.NewPTs();
        for (auto &n : newpts) 
        {
            CalProb(n);
            auto it = priority.begin();
            while (it != priority.end() && it->prob >= n.prob) 
            {
                ++it;
            }
            priority.insert(it, n);
        }
    }
}

// 全局缓冲
static char* global_valbuf = nullptr;
static int*  global_vallens = nullptr;
static char* global_outbuf = nullptr;

// GPU 常量前缀
__device__ __constant__ char d_prefix[64 * MAX_VALUE_LEN];
__device__ __constant__ int  d_prefix_len;

// 并行 Kernel：拼接前缀 + 最后 segment
__global__ void batchLastSegKernel(
    const char* val_buf, const int* val_lens, int total,
    char* out_buf) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;
    int plen = d_prefix_len;
    char* tgt = out_buf + idx * MAX_VALUE_LEN * 2;
    __shared__ char shared_prefix[64 * MAX_VALUE_LEN];
    if (threadIdx.x < plen)
        shared_prefix[threadIdx.x] = d_prefix[threadIdx.x];
    __syncthreads();
    for (int i = 0; i < plen; ++i) tgt[i] = shared_prefix[i];
    int len = val_lens[idx];
    char* src = (char*)(val_buf + idx * MAX_VALUE_LEN);
    for (int i = 0; i < len; ++i) tgt[plen + i] = src[i];
    tgt[plen + len] = '\0';
}

void InitGenerateGPUResources(size_t max_values) {
    hipMalloc(&global_valbuf, max_values * MAX_VALUE_LEN);
    hipMalloc(&global_vallens, max_values * sizeof(int));
    hipMalloc(&global_outbuf, max_values * MAX_VALUE_LEN * 2);
}

// 保持原有Generate签名，使用单次GPU调用生成最后segment
void launchLastSegment(const string &prefix,
                      const vector<string> &values,
                      vector<string> &guesses) 
{
    int total = (int)values.size();
    if (!total) 
    {
        return;
    }

    // 如果未初始化全局缓冲，则按当前total大小初始化一次
    if (global_valbuf == nullptr) 
    {
        InitGenerateGPUResources(total);
    }

    // 拷贝前缀到常量内存
    hipMemcpyToSymbol(HIP_SYMBOL(d_prefix), prefix.c_str(), prefix.size());
    int prelen = (int)prefix.size();
    hipMemcpyToSymbol(HIP_SYMBOL(d_prefix_len), &prelen, sizeof(int));
    
    // 准备值缓冲区
    vector<char> h_valbuf(total * MAX_VALUE_LEN);
    vector<int> h_vallens(total);
    for (int i = 0; i < total; ++i) 
    {
        strncpy(&h_valbuf[i * MAX_VALUE_LEN], values[i].c_str(), MAX_VALUE_LEN);
        h_vallens[i] = values[i].size();
    }
    
    hipMemcpy(global_valbuf, h_valbuf.data(), total * MAX_VALUE_LEN, hipMemcpyHostToDevice);
    hipMemcpy(global_vallens, h_vallens.data(), total * sizeof(int), hipMemcpyHostToDevice);
    
    // 启动GPU Kernel
    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    batchLastSegKernel<<<blocks, threads>>>(global_valbuf, global_vallens, total, global_outbuf);
    hipDeviceSynchronize();
    
    // 将结果从GPU拷贝回主机
    vector<char> h_out(total * MAX_VALUE_LEN * 2);
    hipMemcpy(h_out.data(), global_outbuf, total * MAX_VALUE_LEN * 2, hipMemcpyDeviceToHost);
    for (int i = 0; i < total; ++i)
    {
        guesses.emplace_back(string(&h_out[i * MAX_VALUE_LEN * 2]));
    }
}


// 这个函数是PCFG并行化算法的主要载体，现已优化为GPU并行版本
// 尽量看懂，然后进行并行实现
void PriorityQueue::Generate(PT pt)
{
    // 计算PT的概率，这里主要是给PT的概率进行初始化
    CalProb(pt);

    // 对于只有一个segment的PT，直接遍历生成其中的所有value即可
    if (pt.content.size() == 1)
    {
        // 指向最后一个segment的指针，这个指针实际指向模型中的统计数据
        segment *a;
        // 在模型中定位到这个segment
        if (pt.content[0].type == 1)
        {
            a = &m.letters[m.FindLetter(pt.content[0])];
        }
        if (pt.content[0].type == 2)
        {
            a = &m.digits[m.FindDigit(pt.content[0])];
        }
        if (pt.content[0].type == 3)
        {
            a = &m.symbols[m.FindSymbol(pt.content[0])];
        }
        
        for (int i = 0; i < pt.max_indices[0]; i += 1)
        {
            string guess = a->ordered_values[i];
            // cout << guess << endl;
            guesses.emplace_back(guess);
            total_guesses += 1;
        }
    }
    else
    {
        string prefix;
        
        // 这个for循环的作用：给当前PT的所有segment赋予实际的值（最后一个segment除外）
        // segment值根据curr_indices中对应的值加以确定
        for (int i = 0; i + 1 < pt.curr_indices.size(); ++i) 
        {
            auto &seg = pt.content[i]; 
            int idx = pt.curr_indices[i];
            if (seg.type == 1) 
            {
                prefix += m.letters[m.FindLetter(seg)].ordered_values[idx];
            }
            else if (seg.type == 2) 
            {
                prefix += m.digits[m.FindDigit(seg)].ordered_values[idx];
            }
            else 
            {
                prefix += m.symbols[m.FindSymbol(seg)].ordered_values[idx];
            }
        }

        // 指向最后一个segment的指针，这个指针实际指向模型中的统计数据
        auto &last = pt.content.back();
        segment *a;
        if (last.type == 1)
        {
            a = &m.letters[m.FindLetter(last)];
        }
        else if (last.type == 2)
        {
            a = &m.digits[m.FindDigit(last)];
        }
        else
        {
            a = &m.symbols[m.FindSymbol(last)];
        }
        
        launchLastSegment(prefix, a->ordered_values, guesses);
        total_guesses += a->ordered_values.size();
    }
}

